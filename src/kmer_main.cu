#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <string.h>
#include "tipos.h"
#include "kmer.cuh"

void DeviceInfo(uint8_t device, lint *maxGridSize, lint *maxThreadDim, lint *deviceMemory)
{
   hipDeviceProp_t prop;

   hipGetDeviceProperties(&prop, device);

   printf("\n\n***** Device information *****\n\n");

   printf("\tId: %d\n", device);
   printf("\tName: %s\n", prop.name);
   printf("\tTotal global memory: %ld\n", prop.totalGlobalMem);
   printf("\tMax grid size: %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
   printf("\tMax thread dim: %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
   printf("\tWarp size: %d\n", prop.warpSize);
   printf("\tMax threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);

   printf("\n************************************\n\n");

   *maxThreadDim = prop.maxThreadsDim[0];
   *maxGridSize = prop.maxGridSize[0];
   *deviceMemory = prop.totalGlobalMem;
}

//int main(int argc, char *argv[])
void kmer_main(struct read *rd, lint nN, lint nS, ushort k, ushort device)
{

   int *d_Index;// Index vector
   short *d_Seq;// Seq matrix
   int *Freq, *d_Freq;// Frequence vector
   int fourk;// 4 power k
   int *d_start, *d_length;// The beggining and the length of each sequence
   lint block[3], grid[3];// Grid config; 0:nN, 1:nS
   lint maxGridSize, maxThreadDim, deviceMemory;// Device config
   ushort offset[3] = {1,1,1};
   size_t size[4], totalsize;

   d_Index =  NULL;
   d_Seq = NULL;

   fourk = POW(k);

   hipSetDevice(device);
   DeviceInfo(device, &maxGridSize, &maxThreadDim, &deviceMemory);
   printf("\nnS: %ld, nN: %ld\n", nS, nN);

//---------------------------------------------------------------------------
   size[0] = nN * sizeof(short);// d_Seq and Seq size
   size[1] = nN * sizeof(int); // d_Index and Index size
   size[2] = nS * sizeof(int);  // d_start and d_length
   size[3] = nS * fourk * sizeof(int);// Freq and d_Freq
   totalsize = size[0] + size[1] + (size[2] * 2) + size[3];

   if (totalsize > deviceMemory)
   {
      printf("\n\n\t\t\t[Erro] Nao ha espaco suficiente para alocacao dos dados na gpu\n");
      printf("\t\t\t[Erro] Espaco requerico %ld; Espaco disponivel: %ld\n", totalsize, deviceMemory);
      exit(1);
   }

/*   for (int i = 0; i < nN; i++)
   {
      printf("%d\n", rd->data[i]);
   }
   puts("\nFim");
*/
//---------------------------------------------------------------------------

   if ( hipMalloc    ((void**)&d_Seq, size[0]) != hipSuccess) printf("\nErro1!\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipMalloc    ((void**)&d_Index, size[1]) != hipSuccess) printf("\nErro2!\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipMalloc    ((void**)&d_start, size[2]) != hipSuccess) printf("\nErro3!\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipMalloc    ((void**)&d_length, size[2]) != hipSuccess) printf("\nErro4!\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipHostMalloc((void**)&Freq, size[3]) != hipSuccess) printf("\nErro5!\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipMalloc    ((void**)&d_Freq, size[3]) != hipSuccess) printf("\nErro6!\n");
   puts(hipGetErrorString(hipGetLastError()));

   if ( hipMemset    (d_Freq, 0, size[3]) != hipSuccess) printf("\nErro7\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipMemset    (d_Index, -1, size[3]) != hipSuccess) printf("\nErro8\n");
   puts(hipGetErrorString(hipGetLastError()));

//************************************************
   block[0] = maxThreadDim;
   grid[0] = floor( nN / block[0] );
   if (grid[0] > maxGridSize)
   {
      grid[0] = maxGridSize;
      offset[0] = (nN / (grid[0] * block[0])) + 1;
   }
//   printf("grid: %d\n", grid[0]);
//   printf("block: %d\n", block[0]);

   block[1] = maxThreadDim;
   grid[1] = (nS / block[1]) + 1;
   if (grid[1] > maxGridSize)
   {
      grid[1] = maxGridSize;
      offset[1] = (nS / (grid[1] * block[1])) + 1;
   }
//   printf("grid: %d\n", grid[1]);
//   printf("block: %d\n", block[1]);

   block[2] = maxThreadDim;
   grid[2] = nS;
   if (nS > maxGridSize)
   {
      grid[2] = maxGridSize;
      offset[2] = (nS / grid[2]) + 1;
   }
   printf("grid: %d\n", grid[2]);
   printf("block: %d\n", block[2]);
   printf("offset: %d\n", offset[2]);

//************************************************

   if ( hipMemcpyAsync(d_Seq, rd->data, size[0], hipMemcpyHostToDevice) != hipSuccess) printf("Erro9!\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipMemcpyAsync(d_start, rd->start, size[2], hipMemcpyHostToDevice) != hipSuccess) printf("Erro10!\n");
   puts(hipGetErrorString(hipGetLastError()));
   if ( hipMemcpyAsync(d_length, rd->length, size[2], hipMemcpyHostToDevice) != hipSuccess) printf("Erro11!\n");
   puts(hipGetErrorString(hipGetLastError()));

//************************************************

   //SetMatrix<<<grid[0], block[0]>>>(d_Index, offset[0], -1);
   //SetMatrix<<<nS, POW(k)>>>(d_Freq, offset[0], 0);
   //hipDeviceSynchronize();
   ComputeIndex<<<grid[0], block[0]>>>(d_Seq, d_Index, k, nN, offset[0]);
   puts(hipGetErrorString(hipGetLastError()));
   ComputeFreq<<<grid[1], block[1]>>>(d_Index, d_Freq, d_start, d_length, offset[1], fourk, nS, nN);
   puts(hipGetErrorString(hipGetLastError()));
   //ComputeFreqNew<<<grid[2],block[2]>>>(d_Index, d_Freq, d_start, d_length, offset[2], fourk, nS);

   hipMemcpy(Freq, d_Freq, size[3], hipMemcpyDeviceToHost);
   puts(hipGetErrorString(hipGetLastError()));

   int cont = 0;
   int cont_seq = 0;
   for (int i = 0; i < (nS*fourk); i++)
   {
      if (i % fourk == 0)
      {
         cont = 0;
         printf("%d\n", cont_seq);
         cont_seq++;
      }
      printf("%d: %d\n", cont, Freq[i]);
      cont++;
   }
   printf("\n");

//************************************************
   hipFree(d_Seq);
   hipFree(d_Freq);
   hipFree(d_Index);
   hipFree(d_start);
   hipFree(d_length);
   //hipFree(Freq);
//---------------------------------------------------------------------------

   printf("\nFim kmer_main\n");
}
