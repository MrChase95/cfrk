#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include "kmer.cuh"
#include "tipos.h"
#include "fastaIO.h"

int SelectDevice(int devCount)
{

   int i, device = 0;
   hipDeviceProp_t prop[devCount];

   if (devCount > 0)
   {
      for (i = 0; i < devCount; i++)
      {
         hipGetDeviceProperties(&prop[i], i);
      }

      for (i = 0; i < devCount; i++)
      {
         if (prop[i].totalGlobalMem > prop[device].totalGlobalMem)
         {
            device = i;
         }
      }
   }
   else
      return 0;

return device;
}

struct read* SelectChunk(struct read *rd, ushort chunkSize, ushort it, lint gnS, lint *nS, lint gnN, lint *nN)
{
   struct read *chunk;
   int i;
   lint length = 0;

   // Size to be allocated
   for (i = 0; i < chunkSize; i++)
   {
      int id = chunkSize*it + i;
      if (id > gnS-1)
      {
         break;
      }
      length += rd->length[id]+1;
   }

   hipHostMalloc((void**)&chunk, sizeof(struct read));
   hipHostMalloc((void**)&chunk->data, sizeof(short)*length);
   hipHostMalloc((void**)&chunk->length, sizeof(int)*chunkSize);
   hipHostMalloc((void**)&chunk->start, sizeof(int)*chunkSize);

   // Copy rd->data to chunk->data
   lint start = rd->start[chunkSize*it];
   lint end = start + length;
   for (i = start; i < end; i++)
   {
      chunk->data[i-start] = rd->data[i];
   }

   chunk->length[0] = rd->length[chunkSize*it];
   chunk->start[0] = 0;
   // Copy start and length
   for (i = 1; i < chunkSize; i++)
   {
      int id = chunkSize*it + i;
      chunk->length[i] = rd->length[id];
      chunk->start[i] = chunk->start[i-1]+(chunk->length[i-1]+1);
   }

   *nN = length;
   *nS = chunkSize;
return chunk;
}

int main(int argc, char* argv[])
{

   ushort k;
   int device;
   lint gnN, gnS, nN, nS, chunkSize = 2048;
   int devCount;

   if ( argc < 3)
   {
      printf("Usage: ./kmer [dataset.fasta] [k] <chunkSize: Default 2048>");
      return 1;
   }

   hipDeviceReset();
   
   k = atoi(argv[2]);
   if (argc == 4)
      chunkSize = atoi(argv[3]);

   hipGetDeviceCount(&devCount);
   device = SelectDevice(devCount);

   printf("\ndataset: %s, k: %d, chunkSize: %d\n", argv[1], k, chunkSize);

   lint st = time(NULL);
   puts("\n\n\t\tReading seqs!!!");
   struct read *rd;
   hipHostMalloc((void**)&rd, sizeof(struct read));
   ReadFASTASequences(argv[1], &gnN, &gnS, rd, 1);
   printf("\nnS: %ld, nN: %ld\n", gnS, gnN);
   lint et = time(NULL);

   printf("\n\t\tReading time: %ld\n", (et-st));

   int nChunk = floor(gnS/chunkSize);
   struct read *chunk[nChunk+1];
   for (int i = 0; i < nChunk; i++)
   {
      chunk[i] = SelectChunk(rd, chunkSize, i, gnS, &nS, gnN, &nN);
      kmer_main(chunk[i], nN, nS, k, device);
      //hipDeviceReset();
   }
   int chunkRemain = abs(gnS - (nChunk*chunkSize));
   chunk[nChunk] = SelectChunk(rd, chunkRemain, nChunk, gnS, &nS, gnN, &nN);
   printf("\nnS: %ld, nN: %ld, chunkRemain: %d\n", nS, nN, chunkRemain);
   kmer_main(chunk[nChunk], nN, nS, k, device);

return 0;
}
